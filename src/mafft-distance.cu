
#include <hip/hip_runtime.h>
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#define END_OF_VEC -1

//fill table with values based on pointt
void make_distance_makecompositiontable_p( short *table, int *pointt )
{
	int point;

	while( ( point = *pointt++ ) != END_OF_VEC )
		table[point]++;
}
