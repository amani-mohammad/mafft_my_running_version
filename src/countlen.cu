#include "hip/hip_runtime.h"
//moni Fri Oct 20, 11:21 pm

//this file is called first with input file as parameter. I think it calculates some statistics for input file

#include "mltaln.h"
//#include "io.c"
#include "io.h"

#define DEBUG 0

void countlen_arguments( int argc, char *argv[] )
{
    int c;

    while( --argc > 0 && (*++argv)[0] == '-' )
	{
        while ( (c = *++argv[0]) )
		{
            switch( c )
            {
				case 'i':
					inputfile = *++argv;
//					fprintf( stderr, "inputfile = %s\n", inputfile );
					--argc;
					goto nextoption;
                default:
                    fprintf( stderr, "illegal option %c\n", c );
                    argc = 0;
                    break;
            }
		}
		nextoption:
			;
	}
    if( argc != 0 )
    {
        fprintf( stderr, "options: Check source file !\n" );
        exit( 1 );
    }
}


//int countlen_main( int argc, char *argv[] )
int countlen_main( char * inputFilePath )
{
	FILE *infp;
	int nlenmin;
	double nfreq;

//	countlen_arguments( argc, argv );

	inputfile = inputFilePath;

	if( inputfile )
	{
		printf("input file \n");
		infp = fopen( inputfile, "r" );
		if( !infp )
		{
			fprintf( stderr, "Cannot open %s\n", inputfile );
			exit( 1 );
		}
	}
	else {
		printf("standard input \n");
		infp = stdin;
	}

	//dorp means dna or protein
	dorp = NOTSPECIFIED; //NOTSPECIFIED is a constant in mltaln.h and = 100009
	//this method is in io.c file
	//it reads input file and counts number of sequences in it, frequency of acgt chars, freq of n chars
	//and finds min and max lengths of sequences
	getnumlen_nogap_countn( infp, &nlenmin, &nfreq );

	fprintf( stdout, "%d x %d - %d %c nfreq=%f\n", njob, nlenmax, nlenmin, dorp, nfreq );

	fclose( infp );
	return( 0 );
}
