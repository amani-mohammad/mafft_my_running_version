#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

//#include "countlen.c"
#include "mltaln.h"

#include "countlen.h"
#include "replaceu.h"
#include "makedirectionlist.h"
#include "setdirection.h"

#include "testCuda.h"

//static const int WORK_SIZE = 256;
//
///**
// * This macro checks return value of the CUDA runtime call and exits
// * the application if the call failed.
// */
//#define CUDA_CHECK_RETURN(value) {											\
//	hipError_t _m_cudaStat = value;										\
//	if (_m_cudaStat != hipSuccess) {										\
//		fprintf(stderr, "Error %s at line %d in file %s\n",					\
//				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
//		exit(1);															\
//	} }
//
//__host__ __device__ unsigned int bitreverse(unsigned int number) {
//	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
//	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
//	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
//	return number;
//}
//
///**
// * CUDA kernel function that reverses the order of bits in each element of the array.
// */
//__global__ void bitreverse(void *data) {
//	unsigned int *idata = (unsigned int*) data;
//	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
//}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {

	printf("MAFFT code, ya rab :) \n");

	countlen_main("./src/sample.fa"); //count sequences number, length and DNA or Protein
	printf("done countlen :D \n");

	replaceu_main(dorp, "./src/sample.fa"); //replace unusual characters with X or N based on P or D
	printf("done replace u :D \n");

	//inaccurate direction parameters
//	char* argv[] = {"-F", "-C", "0", "-m", "-I", "0", "-i", "./src/sample.fa", "-t", "0.00", "-r", "5000", "-o", "a"};
	//accurate direction parameters - what I prefer
	char* argv[] = {"-F", "-C", "0", "-m", "-I", "0", "-i", "./src/sample.fa", "-t", "0.00", "-r", "100", "-o", "a", "-d"};
//	freopen("./src/output.txt", "w", stdout);
	make_direction_list_main(15, argv); //make direction list
	fprintf(stderr, "done make direction list :D \n");

	char* argv2[] = {" ", "-d", "./src/direction.txt", "-i", "./src/sample.fa"};
	set_direction_main(5, argv2);
	fprintf(stderr, "done set direction list :D \n");

	main_cuda();

	return 0;
}


//#include <iostream.h>
//
//
//#define CUDA_CHECK_RETURN(value) {											\
//	hipError_t _m_cudaStat = value;										\
//	if (_m_cudaStat != hipSuccess) {										\
//		fprintf(stderr, "Error %s at line %d in file %s\n",					\
//				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
//		exit(1);															\
//	} }
//
//__global__ void add(int a, int b, int *c) {
//	*c = a + b;
//}
//
//int main(void) {
//	int c;
//	int *dev_c;
//
//	CUDA_CHECK_RETURN(hipMalloc((void**) &dev_c, sizeof(int)));
//
//	add<<<1,1>>> (2, 7, dev_c);
//
//	CUDA_CHECK_RETURN(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
//
//	printf("2 + 7 = %d\n", c);
//	hipFree(dev_c);
//
//	hipDeviceProp_t prop;
//	int count;
//
//	CUDA_CHECK_RETURN(hipGetDeviceCount(&count));
//	for (int i = 0;  i < count; ++ i) {
//		CUDA_CHECK_RETURN(hipGetDeviceProperties(&prop, i));
//
//		printf("------ general information for device %d ------ \n", i);
//		printf("Name: %s \n", prop.name);
//		printf("Compute capability: %d.%d \n", prop.major, prop.minor);
//		printf("Clock rate: %d \n", prop.clockRate);
//		printf("Device copy overlap: ");
//		if (prop.deviceOverlap) {
//			printf("Enabled \n");
//		} else {
//			printf("Disabled \n");
//		}
//		printf("Kernel execution timeout: ");
//		if (prop.kernelExecTimeoutEnabled) {
//			printf("Enabled \n");
//		} else {
//			printf("Disabled \n");
//		}
//		printf("----- Memory Information for device %d ----- \n", i);
//		printf("Total global mem: %ld \n", prop.totalGlobalMem);
//		printf("Total constant mem: %ld \n", prop.totalConstMem);
//		printf("Max mem pitch: %ld \n", prop.memPitch);
//		printf("Texture alignment: %ld \n", prop.textureAlignment);
//
//		printf("----- MP Information for device %d ----- \n", i);
//		printf("Multiprocessor count: %d \n", prop.multiProcessorCount);
//		printf("Shared mem per mp: %ld \n", prop.sharedMemPerBlock);
//		printf("Registers per mp: %d \n", prop.regsPerBlock);
//		printf("Threads per wrap: %d \n", prop.warpSize);
//		printf("Max threads per block: %d \n", prop.maxThreadsPerBlock);
//		printf("Max threads dimensions: (%d, %d, %d) \n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
//		printf("Max grid dimensions: (%d, %d, %d) \n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
//		printf("\n");
//
//		printf("----- Other info %d ----- \n", i);
//		printf("Texture pitch alignment: %d \n", prop.texturePitchAlignment);
//		printf("Kernel Execution Timeout Enabled: %d \n", prop.kernelExecTimeoutEnabled);
//		printf("Integrated: %d \n", prop.integrated);
//		printf("Can Map Host Memory: %d \n", prop.canMapHostMemory);
//		printf("Compute mode: %d \n", prop.computeMode);
//		printf("Max Texture 1D: %d \n", prop.maxTexture1D);
//		printf("Surface Alignment: %d \n", prop.surfaceAlignment);
//		printf("Concurrent Kernels: %d \n", prop.concurrentKernels);
//		printf("ECC Enabled: %d \n", prop.ECCEnabled);
//		printf("PCI Bus ID: %d \n", prop.pciBusID);
//		printf("PCI Device ID: %d \n", prop.pciDeviceID);
//		printf("TCC Driver: %d \n", prop.tccDriver);
//		printf("Async Engine Count: %d \n", prop.asyncEngineCount);
//		printf("Unified Addressing: %d \n", prop.unifiedAddressing);
//		printf("Memory Clock Rate: %d \n", prop.memoryClockRate);
//		printf("Global Memory BusWidth: %d \n", prop.memoryBusWidth);
//		printf("L2 Cache Size: %d \n", prop.l2CacheSize);
//		printf("Max Threads Per MultiProcessor: %d \n", prop.maxThreadsPerMultiProcessor);
//		printf("Stream Priorities Supported: %d \n", prop.streamPrioritiesSupported);
//		printf("Global L1 Cache Supported: %d \n", prop.globalL1CacheSupported);
//		printf("Local L1 Cache Supported: %d \n", prop.localL1CacheSupported);
//		printf("Shared Memory Per Multiprocessor: %d \n", prop.sharedMemPerMultiprocessor);
//		printf("Registers Per Multiprocessor: %d \n", prop.regsPerMultiprocessor);
//		printf("Managed Memory: %d \n", prop.managedMemory);
//		printf("Is Multi GPU Board: %d \n", prop.isMultiGpuBoard);
//		printf("Multi GPU Board Group ID: %d \n", prop.multiGpuBoardGroupID);
//
//	}
//
//	return 0;
//}

