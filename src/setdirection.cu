#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

//check directions of nucleotide sequences
#include "mltaln.h"
#include "io.h"

#define DEBUG 0

char *directionfile;
static int show_R_ = 1;

static int subalignment;
static int subalignmentoffset;

void arguments( int argc, char *argv[] )
{
    int c;

	inputfile = NULL;
	directionfile = NULL;
	subalignment = 0;
	subalignmentoffset = 0;
	show_R_ = 1;

    while( --argc > 0 && (*++argv)[0] == '-' )
	{
        while ( (c = *++argv[0]) )
		{
            switch( c )
            {
				case 'd':
					directionfile = *++argv;
					fprintf( stdout, "directionfile = %s\n", directionfile );
					--argc;
					goto nextoption;
				case 'i':
					inputfile = *++argv;
					fprintf( stdout, "inputfile = %s\n", inputfile );
					--argc;
					goto nextoption;
				case 'H':
					subalignment = 1;
					subalignmentoffset = myatoi( *++argv );
					--argc;
					goto nextoption;
				case 'r':
					show_R_ = 0;
					break;
                default:
                    fprintf( stdout, "illegal option %c\n", c );
                    argc = 0;
                    break;
            }
		}
		nextoption:
			;
	}
    if( argc != 0 )
    {
        fprintf( stderr, "options: Check source file !\n" );
        exit( 1 );
    }
}

int set_direction_main( int argc, char *argv[] )
{
	FILE *infp;
	FILE *difp;
	int nlenmin;
	char **name;
	char **seq;
	char *tmpseq;
	char line[100];
	int *nlen;
	int i, j;
	int nsubalignments, maxmem;
	int **subtable = NULL;
	int *preservegaps = NULL;
	char firstdir;
	char *directions;
	FILE *outfp;

	arguments( argc, argv );

	reporterr( "subalignment = %d\n", subalignment );
	reporterr( "subalignmentoffset = %d\n", subalignmentoffset );


	if( inputfile )
	{
		infp = fopen( inputfile, "r" );
		if( !infp )
		{
			fprintf( stderr, "Cannot open %s\n", inputfile );
			exit( 1 );
		}
	}
	else {
		infp = stdin;
	}

	if( directionfile )
	{
		difp = fopen( directionfile, "r" );
		if( !difp )
		{
			fprintf( stderr, "Cannot open %s\n", directionfile );
			exit( 1 );
		}
	}
	else
	{
		fprintf( stderr, "Give directionfile!\n" );
	}

	dorp = NOTSPECIFIED;
	getnumlen_casepreserve( infp, &nlenmin ); //defined in io.c. reads sequences file and set min length in nlenmin and determine dorp value

	fprintf( stdout, "%d x %d - %d %c\n", njob, nlenmax, nlenmin, dorp ); //these values are set in 'getnumlen_casepreserve'

	seq = AllocateCharMtx( njob, nlenmax+1 );
	tmpseq = AllocateCharVec( MAX( B, nlenmax )+1 );
	name = AllocateCharMtx( njob, B+1 );
	nlen = AllocateIntVec( njob );
	directions = (char *) calloc( njob, sizeof( int ) );

	readData_pointer_casepreserve( infp, name, nlen, seq ); //defined in io.c. fill name, seq and nlen arrays with names, sequences and their lengths

	for( i=0; i<njob; i++ )
	{
		fgets( line, 99, difp ); //read line from 'difp' file to 'line' string with max 99 chars
		fprintf( stdout, "line read = %s \n", line);
		if( line[0] != '_' )
		{
			fprintf( stderr, "Format error!\n" );
			exit( 1 );
		}
		if( line[1] == 'R' )
		{
			directions[i] = 'R';
			sreverse( tmpseq, seq[i] ); //defined in io.c. fills tmpseq with reversed chars from seq[i]
			strcpy( seq[i], tmpseq ); //copy reversed seq from tmpseq to seq[i]

			strncpy( tmpseq, name[i]+1, B-3 );
			tmpseq[B-3] = 0;
			if( show_R_ ) //show _R_ at the start of the sequence name
			{
				strcpy( name[i]+1, "_R_" );
				strcpy( name[i]+4, tmpseq );
			}
			else
			{
				strcpy( name[i]+1, tmpseq );
			}
		}
		else if( line[1] == 'F' )
		{
			directions[i] = 'F';
		}
		else
		{
			fprintf( stderr, "Format error!\n" );
			exit( 1 );
		}
	}

	if( subalignment )
	{
		//Set max spaces count to maxmem and number of subalignments in nsubalignments
		readsubalignmentstable( njob, NULL, NULL, &nsubalignments, &maxmem ); //defined in io.c.
		reporterr(       "nsubalignments = %d\n", nsubalignments );
		reporterr(       "maxmem = %d\n", maxmem );
		subtable = AllocateIntMtx( nsubalignments, maxmem+1 );
		preservegaps = AllocateIntVec( njob );
		//checks for sequences existence in multiple groups or sequence number doesn't exist in input file
		readsubalignmentstable( njob, subtable, preservegaps, NULL, NULL );

		for( j=0; j<nsubalignments; j++ ) //check sequences directions and its consistency
		{
			reporterr( "Checking directions of sequences in subalignment%d\n", j );
			firstdir = directions[subtable[j][0]];
			reporterr( "firstdir = %c\n", firstdir );
			for( i=0; subtable[j][i]>-1; i++ )
			{
				if( directions[subtable[j][i]] != firstdir )
				{
					reporterr( "\n\n#############################################################################\n" );
					reporterr( "\nDirection of nucleotide sequences seems to be inconsistent.\n" );
					reporterr( "Please check the following two sequences:\n" );
					reporterr( "	Sequece no.%d (%s)\n", subtable[j][0]+1, name[subtable[j][0]] );
					reporterr( "	Sequece no.%d (%s)\n", subtable[j][i]+1, name[subtable[j][i]] );
					reporterr( "\nThese sequences are in sub alignment no.%d in your setting of --merge,\nbut their directions seem to be different.\n\n", j+1 );
					reporterr( "#############################################################################\n\n\n\n" );
					exit( 1 );
				}
			}
			reporterr( "OK!\n" );
		}
	}

	outfp = fopen("./src/direction_output.txt", "w");

	for( i=0; i<njob; i++ )
	{
		fprintf( stdout, ">%s\n", name[i]+1 ); // > in makefile after the command redirects stdout to the output file
		fprintf( stdout, "%s\n", seq[i] ); //so, this stdout is saved in output file mentioned in makefile

		fprintf( outfp, ">%s\n", name[i]+1 ); // > in makefile after the command redirects stdout to the output file
		fprintf( outfp, "%s\n", seq[i] ); //so, this stdout is saved in output file mentioned in makefile
	}

	fclose(outfp);
	free( nlen );
	FreeCharMtx( seq );
	FreeCharMtx( name );
	free( tmpseq );

	return( 0 );
}
