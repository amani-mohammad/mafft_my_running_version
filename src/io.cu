#include "hip/hip_runtime.h"
//moni Fri Oct 21, 12:21 am

//all files read/write methods and other streams methods
#include "mltaln.h"

static int upperCase = 0;

#define DEBUG   0
#define IODEBUG 0

char creverse( char f )
{
	static TLS char *table = NULL;

	if( f == 0 )
	{
		free( table );
		table = NULL;
		return( 0 );
	}

	if( table == NULL )
	{
		int i;
		table = AllocateCharVec(0x80);
		for( i=0; i<0x80; i++ ) table[i] = i;
		table['A'] = 'T';
		table['C'] = 'G';
		table['G'] = 'C';
		table['T'] = 'A';
		table['U'] = 'A';
		table['M'] = 'K';
		table['R'] = 'Y';
		table['W'] = 'W';
		table['S'] = 'S';
		table['Y'] = 'R';
		table['K'] = 'M';
		table['V'] = 'B';
		table['H'] = 'D';
		table['D'] = 'H';
		table['B'] = 'V';
		table['N'] = 'N';
		table['a'] = 't';
		table['c'] = 'g';
		table['g'] = 'c';
		table['t'] = 'a';
		table['u'] = 'a';
		table['m'] = 'k';
		table['r'] = 'y';
		table['w'] = 'w';
		table['s'] = 's';
		table['y'] = 'r';
		table['k'] = 'm';
		table['v'] = 'b';
		table['h'] = 'd';
		table['d'] = 'h';
		table['b'] = 'v';
		table['n'] = 'n';
//		table['-'] = '-';
//		table['.'] = '.';
//		table['*'] = '*';
	}
	return( table[(int)f] );
}

//fills r with reversed chars from s
void sreverse( char *r, char *s )
{
	r += strlen( s );
	*r-- = 0;
	while( *s )
		*r-- = creverse( *s++ );
//		*r-- = ( *s++ );
}

void seqLower( int nseq, char **seq )
{
	int i, j, len;
	for( i=0; i<nseq; i++ )
	{
		len = strlen( seq[i] );
		for( j=0; j<len; j++ )
			seq[i][j] = tolower( seq[i][j] );
	}
}

///////countlen methods////////
static int countKUorWA( FILE *fp ) //this method counts the number of sequences in FASTA formatted input file
{
	int value;
	int c, b;

	value= 0;
	b = '\n';
	while( ( c = getc( fp ) ) != EOF )
	{
		if( b == '\n' && ( c == '>' ) ) { //if new sequence, then count up
//			printf("new sequence \n");
			value++;
		}
		b = c;
	}
	rewind( fp ); //reset the file position to the beginning of the input stream
	return( value );
}

void searchKUorWA( FILE *fp )
{
	int c, b;
	b = '\n';
	//reads characters till reach first sequence, then stop
	while( !( ( ( c = getc( fp ) ) == '>' || c == EOF ) && b == '\n' ) )
		b = c;
	ungetc( c, fp ); //pushes current character again to input stream to be available for next 'getc'
}

//what i understand till now is that this reads first line of each sequence(seq. name) into char sequence
//char s[]; int l; FILE *fp;
int myfgets(char s[], int l, FILE* fp)
{
        int     c = 0, i = 0 ;

		if( feof( fp ) ) return( 1 ); //test end of file

		for( i=0; i<l && ( c=getc( fp ) ) != '\n'; i++ ) {
        	*s++ = c;
		}
        *s = '\0' ;
		if( c != '\n' ) {
			while( getc(fp) != '\n' )
				;
		}
		return( 0 );
}

void kake2hiku( char *str )
{
	do
		if( *str == '*' ) *str = '-';
	while( *str++ );
}

int charfilter( unsigned char *str )
{
	unsigned char tmp;
	unsigned char *res = str;
	unsigned char *bk = str;

	while( (tmp=*str++) )
	{
//		if( tmp == '=' || tmp == '*' || tmp == '<' || tmp == '>' || tmp == '(' || tmp == ')' )
		if( tmp == '=' || tmp == '<' || tmp == '>' )
		{
			fprintf( stderr, "\n" );
			fprintf( stderr, "Characters '= < >' are not accepted in the --text mode, \nalthough most printable characters are ok.\n" );
			fprintf( stderr, "\n" );
			exit( 1 );
		}
//		if( 0x20 < tmp && tmp < 0x7f )
//		if( 0x0 <=tmp && tmp < 0x100 &&
		if( tmp != 0x0a && tmp != 0x20 && tmp != 0x0d )
//		if( tmp != '\n' && tmp != ' ' && tmp != '\t' ) // unprintable characters mo ok.
		{
			*res++ = tmp;
//			reporterr( "tmp=%d (%c)\n", tmp, tmp );
		}
	}
	*res = 0;
	return( res - bk );
}

int onlyAlpha_lower( char *str )
{
	char tmp;
	char *res = str;
	char *bk = str;

	while( (tmp=*str++) )
		if( isalpha( tmp ) || tmp == '-' || tmp == '*' || tmp == '.' )
			*res++ = tolower( tmp );
	*res = 0;
	return( res - bk );
}

int onlyAlpha_upper( char *str )
{
	char tmp;
	char *res = str;
	char *bk = str;

	while( (tmp=*str++) )
		if( isalpha( tmp ) || tmp == '-' || tmp == '*' || tmp == '.' )
			*res++ = toupper( tmp );
	*res = 0;
	return( res - bk );
}

char *load1SeqWithoutName_realloc( FILE *fpp )
{
	int c, b;
	char *cbuf;
	int size = N;
	char *val;

	val = (char *)malloc( (size+1) * sizeof( char ) );
	cbuf = val; //point to start of buffer

	b = '\n';
	while( ( c = getc( fpp ) ) != EOF &&
          !( ( c == '>' || c == EOF ) && b == '\n' ) )
	{
		*cbuf++ = (char)c;
		if( cbuf - val == size ) //if buffer exceeds size, double size
		{
			size += N;
			fprintf( stderr, "reallocating...\n" );
			val = (char *)realloc( val, (size+1) * sizeof( char ) );
			if( !val )
			{
				fprintf( stderr, "Allocation error in load1SeqWithoutName_realloc \n" );
				exit( 1 );
			}
			fprintf( stderr, "done.\n" );
			cbuf = val + size-N;
		}
		b = c;
	}
	ungetc( c, fpp );
	*cbuf = 0;

	if( nblosum == -2 )
	{
		charfilter( (unsigned char *) val ); //filter characters in sequence
	}
	else
	{
		if( dorp == 'd' )
			onlyAlpha_lower( val );
		else
			onlyAlpha_upper( val );
		kake2hiku( val );
	}
	return( val );
}

int countnogaplen( char *seq )
{
	int val = 0;
	while( *seq )
		if( *seq++ != '-' ) val++;
	return( val );
}

int countATGCandN( char *s, int *countN, int *total )
{
	int nATGC;
	int nChar;
	int nN;
	char c;
	nN = nATGC = nChar = 0;

	if( *s == 0 )
	{
		*total = 0;
		return( 0 );
	}

	do
	{
		c = tolower( *s ); //convert character to lower case
		if( isalpha( c ) ) //is this character is alphabetic
		{
			nChar++; //increase number of characters by 1
			if( c == 'a' || c == 't' || c == 'g' || c == 'c' || c == 'u' || c == 'n' )
				nATGC++;
			if( c == 'n' )
				nN++;
		}
	}
	while( *++s );

//	reporterr( "nN = %d", nN );

	*total = nChar;
	*countN = nN;
	return( nATGC );
}

int countATGC( char *s, int *total )
{
	int nATGC;
	int nChar;
	char c;
	nATGC = nChar = 0;

	if( *s == 0 )
	{
		*total = 0;
		return( 0 );
	}

	do
	{
		c = tolower( *s );
		if( isalpha( c ) )
		{
			nChar++;
			if( c == 'a' || c == 't' || c == 'g' || c == 'c' || c == 'u' || c == 'n' )
				nATGC++;
		}
	}
	while( *++s );

	*total = nChar;
	return( nATGC );
}

//char *AllocateCharVec( int l1 )
//{
//	char *cvec;
//
//	cvec = (char *)calloc( l1, sizeof( char ) );
//	if( cvec == NULL )
//	{
//		fprintf( stderr, "Cannot allocate %d character vector.\n", l1 );
//		exit( 1 );
//	}
//	return( cvec );
//}


//it reads input file and counts number of sequences in it, frequency of acgt chars, freq of n chars
//and finds min and max lengths of sequences
void getnumlen_nogap_countn( FILE *fp, int *nlenminpt, double *nfreq )
{
	printf("getnumlen_nogap_countn \n");

	int total;
	int nsite = 0;
	int atgcnum, nnum, nN;
	int i, tmp;
	char *tmpseq, *tmpname;
	double atgcfreq;
	tmpname = AllocateCharVec( N ); //N = 5,000,000
	njob = countKUorWA( fp ); //njob = number of sequences in the input file - this var is defined in defs.h
	searchKUorWA( fp ); //this method locates the stream pointer to start of first sequence
	nlenmax = 0; //this var is defined in defs.h
	*nlenminpt = 99999999;
	atgcnum = 0;
	total = 0;
	nnum = 0;

	printf("number of jobs = %d\n", njob);

	for( i=0; i<njob; i++ )
	{
		myfgets( tmpname, N-1, fp ); //read sequence name in tmpname
		tmpseq = load1SeqWithoutName_realloc( fp ); //load sequence characters in tmpseq
		tmp = countnogaplen( tmpseq ); //get count of characters in sequence - without gaps
		if( tmp > nlenmax ) nlenmax  = tmp; //set max sequence length
		if( tmp < *nlenminpt ) *nlenminpt  = tmp; //set min sequence length
		atgcnum += countATGCandN( tmpseq, &nN, &nsite ); //finds number of cgtanu chars, n chars and total chars in sequence
		total += nsite; //total = total num of chars in all sequences
		nnum += nN; //nnum = number of n chars in all sequences
		free( tmpseq ); //free sequence memory
	}
	free( tmpname ); //free sequence name memory
	atgcfreq = (double)atgcnum / total; //get atgc freq in all sequences
	*nfreq = (double)nnum / atgcnum; //get n freq in all sequences
//	fprintf( stderr, "##### nnum = %d\n", nnum );
//	fprintf( stderr, "##### atgcfreq = %f, *nfreq = %f\n", atgcfreq, *nfreq );
	if( dorp == NOTSPECIFIED )
	{
		if( atgcfreq > 0.75 ) //if atgc freq is > 0.75, then dorp is d (dna)
		{
			dorp = 'd';
			upperCase = -1;
		}
		else                  //else, dorp is p (protein)
		{
			dorp = 'p';
			upperCase = 0;
		}
	}
}

//Finds sequences count, max length and dna or protein from fp file
void getnumlen( FILE *fp )
{
	int total;
	int nsite = 0;
	int atgcnum;
	int i, tmp;
	char *tmpseq;
	char *tmpname;
	double atgcfreq;
	tmpname = AllocateCharVec( N ); //N is defined in mltaln.h and = 5,000,000
	njob = countKUorWA( fp ); //number of sequences. defined in defs.h
	searchKUorWA( fp ); //stop at first sequence
	nlenmax = 0; //defined in defs.h
	atgcnum = 0;
	total = 0;
	for( i=0; i<njob; i++ )
	{
		myfgets( tmpname, N-1, fp ); //read sequence name into tmpname
		tmpseq = load1SeqWithoutName_realloc( fp ); //read sequence itself
		tmp = strlen( tmpseq ); //get length of tmpseq
		if( tmp > nlenmax ) nlenmax  = tmp; //save max length of sequences
		atgcnum += countATGC( tmpseq, &nsite ); //count atgc chars in all sequences
		total += nsite; //count total number of chars in all sequences
//		fprintf( stderr, "##### total = %d\n", total );
		free( tmpseq );
	}


	atgcfreq = (double)atgcnum / total;
//	fprintf( stderr, "##### atgcfreq = %f\n", atgcfreq );
	if( dorp == NOTSPECIFIED ) //dorp defined in defs.c
	{
		if( atgcfreq > 0.75 )
		{
			dorp = 'd';
			upperCase = -1; //defined here in io.c
		}
		else
		{
			dorp = 'p';
			upperCase = 0;
		}
	}
	free( tmpname );
}

//It reads sequences and their names from fp file into seq, name and nlen arrays.
void readData_pointer( FILE *fp, char **name, int *nlen, char **seq )
{
	int i;
	static char *tmpseq = NULL;

#if 0
	if( !tmpseq )
	{
		tmpseq = AllocateCharVec( N );
	}
#endif

	rewind( fp ); //point to first character in the fp stream
	searchKUorWA( fp ); //locates the stream pointer to start of first sequence

	for( i=0; i<njob; i++ )
	{
		name[i][0] = '='; getc( fp );
#if 0
		fgets( name[i]+1, B-2, fp );
		j = strlen( name[i] );
		if( name[i][j-1] != '\n' )
			ErrorExit( "Too long name\n" );
		name[i][j-1] = 0;
#else
		myfgets( name[i]+1, B-2, fp ); //read sequence name into 'name[i]' with max length B-2
#endif
#if 0
		fprintf( stderr, "name[%d] = %s\n", i, name[i] );
#endif
		tmpseq = load1SeqWithoutName_realloc( fp ); //load sequence characters in tmpseq
		strcpy( seq[i], tmpseq ); //copy tmpseq to seq[i]
		free( tmpseq );
		nlen[i] = strlen( seq[i] ); //save length of seq[i] in nlen[i]
	}
	if( dorp == 'd' && upperCase != -1 ) seqLower( njob, seq ); //set all chars to lower case
#if 0
	free( tmpseq );
#endif
	if( outnumber ) //outnumber is defined in defs.c, and = 0
	{
		char *namebuf;
		char *cptr;
		namebuf = (char *) calloc( B+100, sizeof( char ) );
		for( i=0; i<njob; i++ )
		{
			namebuf[0] = '=';
			cptr = strstr( name[i], "_numo_e_" ); //find the first occurrence of second param in the name[i]
			if( cptr ) //sprintf send formatted output to first parameter string
				sprintf( namebuf+1, "_numo_s_%08d_numo_e_%s", i+1, cptr+8 );
			else
				sprintf( namebuf+1, "_numo_s_%08d_numo_e_%s", i+1, name[i]+1 );
			strncpy( name[i], namebuf, B ); //copy B chars from namebuf to name[i]
			name[i][B-1] = 0; //add null char at the end of name[i]
		}
		free( namebuf );
//		exit( 1 );
	}
}


////////replaceu methods////////////
char *load1SeqWithoutName_realloc_casepreserve( FILE *fpp )
{
	int c, b;
	char *cbuf;
	int size = N;
	char *val;

	val = (char *) malloc( (size+1) * sizeof( char ) );
	cbuf = val;

	b = '\n';
	while( ( c = getc( fpp ) ) != EOF &&
          !( ( c == '>' || c == EOF ) && b == '\n' ) )
	{
		*cbuf++ = (char)c;
		if( cbuf - val == size )
		{
			size += N;
			fprintf( stderr, "reallocating...\n" );
			val = (char *)realloc( val, (size+1) * sizeof( char ) );
			if( !val )
			{
				fprintf( stderr, "Allocation error in load1SeqWithoutName_realloc \n" );
				exit( 1 );
			}
			fprintf( stderr, "done.\n" );
			cbuf = val + size-N;
		}
		b = c;
	}
	ungetc( c, fpp );
	*cbuf = 0;
//	onlyGraph( val );
	charfilter( (unsigned char *) val );
//	kake2hiku( val );
	return( val );
}

//read sequences in input file, count their number and get cgta freq to determine dorp value
void getnumlen_casepreserve( FILE *fp, int *nlenminpt )
{
	int total;
	int nsite = 0;
	int atgcnum;
	int i, tmp;
	char *tmpseq, *tmpname;
	double atgcfreq;
	tmpname = AllocateCharVec( N ); //allocate memory for sequences names
	njob = countKUorWA( fp ); //get number of sequences in input file
	searchKUorWA( fp ); //point to first sequence name
	nlenmax = 0;
	*nlenminpt = 99999999;
	atgcnum = 0;
	total = 0;
	for( i=0; i<njob; i++ )
	{
		myfgets( tmpname, N-1, fp ); //read sequence name
		tmpseq = load1SeqWithoutName_realloc_casepreserve( fp ); //read sequence chars without changing case
		tmp = strlen( tmpseq ); //length of sequence
		if( tmp > nlenmax ) nlenmax  = tmp;
		if( tmp < *nlenminpt ) *nlenminpt  = tmp;
		atgcnum += countATGC( tmpseq, &nsite ); //count acgt chars in sequence
		total += nsite; //count total chars in all sequences
		free( tmpseq );
	}
	free( tmpname );
	atgcfreq = (double)atgcnum / total; //get acgt chars frequency in total chars count
//	fprintf( stderr, "##### atgcfreq = %f\n", atgcfreq );
	if( dorp == NOTSPECIFIED ) //dna or protein
	{
		if( atgcfreq > 0.75 ) //dna
		{
			dorp = 'd';
			upperCase = -1;
		}
		else                  //protein
		{
			dorp = 'p';
			upperCase = 0;
		}
	}
}

//fill matrices of sequences, sequences names and lengths
void readData_pointer_casepreserve( FILE *fp, char **name, int *nlen, char **seq )
{
	int i;
	static char *tmpseq = NULL;

#if 0
	if( !tmpseq )
	{
		tmpseq = AllocateCharVec( N );
	}
#endif

	rewind( fp );
	searchKUorWA( fp ); //point to first sequence name

	for( i=0; i<njob; i++ )
	{
		name[i][0] = '='; getc( fp );
#if 0
		fgets( name[i]+1, B-2, fp );
		j = strlen( name[i] );
		if( name[i][j-1] != '\n' )
			ErrorExit( "Too long name\n" );
		name[i][j-1] = 0;
#else
		myfgets( name[i]+1, B-2, fp ); //read sequence name into name[i][...]
#endif
#if 0
		fprintf( stderr, "name[%d] = %s\n", i, name[i] );
#endif
		tmpseq = load1SeqWithoutName_realloc_casepreserve( fp ); //read sequence in tmpseq
		strcpy( seq[i], tmpseq ); //then copy to seq[i]
		free( tmpseq );
		nlen[i] = strlen( seq[i] ); //set length of sequence to nlen[i]
	}
}

static void tab2space( char *s ) // nen no tame  //converts tap to space
{
	while( *s )
	{
		if( *s == '\t' ) *s = ' ';
		s++;
	}
}

static int readasubalignment( char *s, int *t, int *preservegaps )
{
	int v = 0;
	char status = 's';
	char *pt = s;
	*preservegaps = 0;
	tab2space( s ); //convert tabs to space
	while( *pt )
	{
		if( *pt == ' ' )
		{
			status = 's';
		}
		else
		{
			if( status == 's' )
			{
				if( *pt == '\n' || *pt == '#' ) break; //exit while loop
				status = 'n';
				t[v] = atoi( pt ); //convert char in pt to int and assign to t[v]
				if( t[v] == 0 )
				{
					fprintf( stderr, "Format error? Sequences must be specified as 1, 2, 3...\n" );
					exit( 1 );
				}
				if( t[v] < 0 ) *preservegaps = 1; //if negative number, set preservegaps to 1
				t[v] = abs( t[v] );
				t[v] -= 1;
				v++;
			}
		}
		pt++;
	}
	t[v] = -1;
	return( v );
}

static int countspace( char *s )
{
	int v = 0;
	char status = 's';
	char *pt = s;
	tab2space( s ); //defined here. converts all taps in s to spaces
	while( *pt )
	{
		if( *pt == ' ' )
		{
			status = 's';
		}
		else
		{
			if( status == 's' )
			{
				if( *pt == '\n' || *pt == '#' ) break; //exit from while loop
				v++;
				status = 'n';
				if( atoi( pt ) == 0 )
				{
					fprintf( stderr, "Format error? Sequences should be specified as 1, 2, 3...\n" );
					exit( 1 );
				}
			}
		}
		pt++;
	}
	return( v );
}

//First call of this method with table = NULL reads number of subalignments in subalignments file and assign to nsubpt
//also reads max number of spaces in all sequences into maxmempt
//Second call reads data from the file and fills table with it
void readsubalignmentstable( int nseq, int **table, int *preservegaps, int *nsubpt, int *maxmempt ) {
	FILE *fp;
	char *line;
	int linelen = 1000000;
	int nmem;
	int lpos;
	int i, p;
	int *tab01;

	line = (char *) calloc( linelen, sizeof( char ) );
	fp = fopen( "_subalignmentstable", "r" ); //I need to know where this file exists and what is its content?
	if( !fp )
	{
		fprintf( stderr, "Cannot open _subalignmentstable\n" );
		exit( 1 );
	}
	if( table == NULL )
	{
		*nsubpt = 0;
		*maxmempt = 0;
		while( 1 )
		{
			fgets( line, linelen-1, fp );
			if( feof( fp ) ) break;
			if( line[strlen(line)-1] != '\n' ) //line length exceeds max length
			{
				fprintf( stderr, "too long line? \n" );
				exit( 1 );
			}
			if( line[0] == '#' ) continue; //comment line, so jump to next iteration, i. e. line
			if( atoi( line ) == 0 ) continue; //jump to next iteration, i. e. line
			nmem = countspace( line ); //defined here. converts all tabs in line to spaces and counts their number
			if( nmem > *maxmempt ) *maxmempt = nmem; //maxmempt contains max number of spaces in all subalignments
			(*nsubpt)++; //increment count of subalignments
		}
	}
	else
	{
		tab01 = (int *) calloc( nseq, sizeof( int ) );
		for( i=0; i<nseq; i++ ) tab01[i] = 0;
		lpos = 0;
		while( 1 )
		{
			fgets( line, linelen-1, fp );
			if( feof( fp ) ) break;
			if( line[strlen(line)-1] != '\n' ) //line length exceeds max length
			{
				fprintf( stderr, "too long line? \n" );
				exit( 1 );
			}
			if( line[0] == '#' ) continue; //comment line, so jump to next iteration, i. e. line
			if( atoi( line ) == 0 ) continue; //jump to next iteration, i. e. line
			readasubalignment( line, table[lpos], preservegaps+lpos ); //defined here. read sequence in line and fill table[lpos] with chars
			for( i=0; (p=table[lpos][i])!=-1; i++ ) //i think this loop checks for duplicated sequences in different groups
			{
				if( tab01[p] )
				{
					fprintf( stderr, "\nSequence %d appears in different groups.\n", p+1 );
					fprintf( stderr, "Hierarchical grouping is not supported.\n\n" );
					exit( 1 );
				}
				tab01[p] = 1;
				if( p > nseq-1 )
				{
					fprintf( stderr, "Sequence %d does not exist in the input sequence file.\n", p+1 );
					exit( 1 );
				}
			}
			lpos++;
		}
		free( tab01 );
	}
	fclose( fp );
	free( line );
}


void ErrorExit( char *message )
{
	fprintf( stderr, "%s\n", message );
	exit( 1 );
}

//inits signalSM value which is defined in defs.h.
void initSignalSM( void )
{
//	int signalsmid;

#if IODEBUG
	if( ppid ) fprintf( stderr, "PID of xced = %d\n", ppid );
#endif
	if( !ppid ) //ppid is int defined in defs.h
	{
		signalSM = NULL; //signalSM is char* defined in defs.h
		return;
	}

#if 0
	signalsmid = shmget( (key_t)ppid, 3, IPC_ALLOC | 0666 );
	if( signalsmid == -1 ) ErrorExit( "Cannot get Shared memory for signal.\n" );
	signalSM = shmat( signalsmid, 0, 0 );
	if( (int)signalSM == -1 ) ErrorExit( "Cannot attatch Shared Memory for signal!\n" );
	signalSM[STATUS] = IMA_KAITERU;
	signalSM[SEMAPHORE] = 1;
#endif
}

//init prep_g and trap_g files. I think these files are for tracing
void initFiles( void )
{
	char pname[100];
	if( ppid )
		sprintf( pname, "/tmp/pre.%d", ppid );
	else
		sprintf( pname, "pre" );
	prep_g = fopen( pname, "w" ); //prep_g is FILE* defined in defs.h
	if( !prep_g ) ErrorExit( "Cannot open pre" );

	trap_g = fopen( "trace", "w" ); //trap_g is FILE* defined in defs.h
	if( !trap_g ) ErrorExit( "cannot open trace" );
	fprintf( trap_g, "PID = %d\n", getpid() ); //getpid -> get process ID
	fflush( trap_g );
}

void closeFiles( void )
{
	fclose( prep_g );
	fclose( trap_g );
}

static void showaamtxexample()
{
	fprintf( stderr, "Format error in aa matrix\n" );
	fprintf( stderr, "# Example:\n" );
	fprintf( stderr, "# comment\n" );
	fprintf( stderr, "   A  R  N  D  C  Q  E  G  H  I  L  K  M  F  P  S  T  W  Y  V\n" );
	fprintf( stderr, "A  4 -1 -2 -2  0 -1 -1  0 -2 -1 -1 -1 -1 -2 -1  1  0 -3 -2  0\n" );
	fprintf( stderr, "R -1  5  0 -2 -3  1  0 -2  0 -3 -2  2 -1 -3 -2 -1 -1 -3 -2 -3\n" );
	fprintf( stderr, "...\n" );
	fprintf( stderr, "V  0 -3 -3 -3 -1 -2 -2 -3 -3  3  1 -2  1 -1 -2 -2  0 -3 -1  4\n" );
	fprintf( stderr, "frequency 0.07 0.05 0.04 0.05 0.02 .. \n" );
	fprintf( stderr, "# Example end\n" );
	fprintf( stderr, "Only the lower half is loaded\n" );
	fprintf( stderr, "The last line (frequency) is optional.\n" );
	exit( 1 );
}

double *loadaamtx( void ) //called when Blosum number = -1 ---- read user defined matrix and return it
{
	int i, j, k, ii, jj;
	double *val;
	double **raw;
	int *map;
	char *aaorder = "ARNDCQEGHILKMFPSTWYV";
	char *inorder;
	char *line;
	char *ptr1;
	char *ptr2;
	char *mtxfname = "_aamtx";
	FILE *mf;

	raw = AllocateDoubleMtx( 21, 20 );
	val = AllocateDoubleVec( 420 );
	map = AllocateIntVec( 20 );

	if( dorp != 'p' )
	{
		fprintf( stderr, "User-defined matrix is not supported for DNA\n" );
		exit( 1 );
	}

	mf = fopen( mtxfname, "r" );
	if( mf == NULL )
	{
		fprintf( stderr, "Cannot open the _aamtx file\n" );
		exit( 1 );
	}

	inorder = (char *) calloc( 1000, sizeof( char ) );
	line = (char *) calloc( 1000, sizeof( char ) );


	while( !feof( mf ) )
	{
		fgets( inorder, 999, mf );
		if( inorder[0] != '#' ) break;
	}
	ptr1 = ptr2 = inorder;
	while( *ptr2 )
	{
		if( isalpha( *ptr2 ) )
		{
			*ptr1 = toupper( *ptr2 );
			ptr1++;
		}
		ptr2++;
	}
	inorder[20] = 0;

	for( i=0; i<20; i++ )
	{
		ptr2 = strchr( inorder, aaorder[i] );
		if( ptr2 == NULL )
		{
			fprintf( stderr, "%c: not found in the first 20 letters.\n", aaorder[i] );
			showaamtxexample(); //defined here in io.c. Shows error and example for aamtx then exit
		}
		else
		{
			map[i] = ptr2 - inorder;
		}
	}

	i = 0;
	while( !feof( mf ) )
	{
		fgets( line, 999, mf );
//		fprintf( stderr, "line = %s\n", line );
		if( line[0] == '#' ) continue;
		ptr1 = line;
//		fprintf( stderr, "line = %s\n", line );
		for( j=0; j<=i; j++ )
		{
			while( !isdigit( *ptr1 ) && *ptr1 != '-' && *ptr1 != '.' )
				ptr1++;

			raw[i][j] = atof( ptr1 );
//			fprintf( stderr, "raw[][]=%f, %c-%c %d-%d\n", raw[i][j], inorder[i], inorder[j], i, j );
			ptr1 = strchr( ptr1, ' ' );
			if( ptr1 == NULL && j<i) showaamtxexample();
		}
		i++;
		if( i > 19 ) break;
	}

	for( i=0; i<20; i++ ) raw[20][i] = -1.0;
	while( !feof( mf ) )
	{
		fgets( line, 999, mf );
		if( line[0] == 'f' )
		{
//			fprintf( stderr, "line = %s\n", line );
			ptr1 = line;
			for( j=0; j<20; j++ )
			{
				while( !isdigit( *ptr1 ) && *ptr1 != '-' && *ptr1 != '.' )
					ptr1++;

				raw[20][j] = atof( ptr1 );
//				fprintf( stderr, "raw[20][]=%f, %c %d\n", raw[20][j], inorder[i], j );
				ptr1 = strchr( ptr1, ' ' );
				if( ptr1 == NULL && j<19) showaamtxexample();
			}
			break;
		}
	}

	k = 0;
	for( i=0; i<20; i++ )
	{
		for( j=0; j<=i; j++ )
		{
			if( i != j )
			{
				ii = MAX( map[i], map[j] );
				jj = MIN( map[i], map[j] );
			}
			else ii = jj = map[i];
			val[k++] = raw[ii][jj];
//			fprintf( stderr, "%c-%c, %f\n", aaorder[i], aaorder[j], val[k-1] );
		}
	}
	for( i=0; i<20; i++ ) val[400+i] = raw[20][map[i]];

	fprintf( stderr, "inorder = %s\n", inorder );
	fclose( mf );
	free( inorder );
	free( line );
	FreeDoubleMtx( raw );
	free( map );
	return( val );
}

char *progName( char *str )
{
    char *value;
    if( ( value = strrchr( str, '/' ) ) != NULL )
        return( value+1 );
    else
        return( str );
}

int myatoi( char *in )
{
	if( in == NULL )
	{
		fprintf( stderr, "Error in myatoi()\n" );
		exit( 1 );
	}
	return( atoi( in ) ); //'atoi' converts the string argument to integer
}

void reporterr( const char *str, ... )
{
//	static int loglen = 0;
	va_list args;

	if( gmsg )
	{
# if 1  // ato de sakujo
		static FILE *errtmpfp = NULL;
		if( errtmpfp == NULL )
			errtmpfp = fopen( "maffterr", "w" );
		else
			errtmpfp = fopen( "maffterr", "a" );
		va_start( args, str );
		vfprintf( errtmpfp, str, args );
		va_end( args );
		fclose( errtmpfp );
#endif

#if 0
		char *tmpptr;
		tmpptr = (char *)realloc( *gmsg, (loglen+10000) * sizeof( char ) );
		if( tmpptr == NULL )
		{
			fprintf( stderr, "Cannot relloc *gmsg\n" );
			exit( 1 );
		}
		*gmsg = tmpptr;
		va_start( args, str );
		loglen += vsprintf( *gmsg + loglen, str, args );
		va_end( args );


		va_start( args, str );
		loglen += vsprintf( *gmsg + loglen, str, args );
		va_end( args );
		*(*gmsg + loglen) = 0;
		if( loglen > gmsglen - 100 ) loglen = 0; // tekitou
#endif

	}
	else
	{
		va_start( args, str );
		vfprintf( stderr, str, args );
		va_end( args );
//		fflush( stderr ); // iru?
	}
	return;
}

